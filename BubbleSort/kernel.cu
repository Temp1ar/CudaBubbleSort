#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <stdio.h>
#include <vector>
#include <limits>
#include <algorithm>

#include <Windows.h>

hipError_t sortWithCuda(int *a, size_t size, float* time);

typedef long long int64; 
typedef unsigned long long uint64;
__host__ int64 GetTimeMs64()
{
	 /* Windows */
	 FILETIME ft;
	 LARGE_INTEGER li;

	 /* Get the amount of 100 nano seconds intervals elapsed 
	  * since January 1, 1601 (UTC) and copy it
	  * to a LARGE_INTEGER structure. */
	 GetSystemTimeAsFileTime(&ft);
	 li.LowPart = ft.dwLowDateTime;
	 li.HighPart = ft.dwHighDateTime;

	 uint64 ret = li.QuadPart;
	 ret -= 116444736000000000LL; 
	 /* Convert from file time to UNIX epoch time. */
	 ret /= 10000; /* From 100 nano seconds (10^-7) 
				   to 1 millisecond (10^-3) intervals */

	 return ret;
}

__global__ void swapOnKernel(int *a, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x * 2;
	int cacheFirst;
	int cacheSecond;
	int cacheThird;

    for (int j = 0; j < size/2 + 1; j++) {

	    if(i+1 < size) {
		    cacheFirst = a[i];
		    cacheSecond = a[i+1];

		    if(cacheFirst > cacheSecond) {
			    int temp = cacheFirst;
			    a[i] = cacheSecond;
			    cacheSecond = a[i+1] = temp;
		    }
	    }

	    if(i+2 < size) {
		    cacheThird = a[i+2];
		    if(cacheSecond > cacheThird) {
			    int temp = cacheSecond;
			    a[i+1] = cacheThird;
			    a[i+2] = temp;
		    }
	    }

        __syncthreads();
    }

}

__host__ void bubbleSort(int arr[], int n) {
	// algorithm from http://www.algolist.net/Algorithms/Sorting/Bubble_sort
	bool swapped = true;
	int j = 0;
	int tmp;
	while (swapped) {
		swapped = false;
		j++;
		for (int i = 0; i < n - j; i++) {
			if (arr[i] > arr[i + 1]) {
				tmp = arr[i];
				arr[i] = arr[i + 1];
				arr[i + 1] = tmp;
				swapped = true;
			}
		}
	}
}

int main()
{
	srand((unsigned)time(0)); 
    const int arraySize = 512;

	// Create vector and fill it with values
	std::vector<int> a(arraySize);
	for (int i = 0; i < arraySize; ++i) {
		a[i] = 100000000 + arraySize - i;//(rand() * (rand() - 1));//
	}
	std::vector<int> b(a);
	
	float time = 0.0;
    // Swap elements in parallel.
    hipError_t cudaStatus = sortWithCuda(&a[0], a.size(), &time);

	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sortWithCuda failed!");
        return 1;
    }

	bool sortingSuccessful = true;
	for (int i = 0; i < a.size()-1; ++i) {
		if (a[i] > a[i+1]) {
			sortingSuccessful = false;
			break;
		}
		// printf("%d, ", a[i]);
	}
	printf("\n");

	printf ("Time for the GPU: %f ms\n", time);

	if(!sortingSuccessful) {
		printf("Sorting failed.\n");
	}

	int64 stlSortStart = GetTimeMs64();
	bubbleSort(&b[0], b.size());
	int64 stlSortFinish = GetTimeMs64();
	printf ("Time for the CPU: %d ms\n", 
		(stlSortFinish - stlSortStart));


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	//getchar();

    return 0;
}

// Helper function for using CUDA to sort vectors in parallel.
__host__ hipError_t sortWithCuda(int *a, size_t size, float* time)
{
    int *dev_a = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for one vectors.
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

     // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Create timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
    // Launch a kernel on the GPU with one thread for each element.
    swapOnKernel<<<1, size/2>>>(dev_a, size);

    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(time, start, stop);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    
    return cudaStatus;
}
